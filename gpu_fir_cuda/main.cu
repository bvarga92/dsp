#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "fir.cuh"

void firCPU(float *input, float *output, float *coeffs, int samples, int taps){
	clock_t s0, e0;
	s0=clock();
	for(int i=0;i<CPU_RUNS;i++){
		for(int out_addr=0;out_addr<SAMPLE_NUM;out_addr++){
			float acc=0.0f;
			for(int tap=taps-1;tap>=0;tap--){
				int in_addr=out_addr-tap;
				if(in_addr>=0) acc+=coeffs[tap]*input[in_addr];
			}
			output[out_addr]=acc;
		}
	}
	e0=clock();
	printf("Execution time with CPU: %.1f us\n",(double)(e0-s0)*1000000.0/(CLOCKS_PER_SEC*CPU_RUNS));
}

void firCUDA(float *input, float *output, float *coeffs, int samples, int taps, bool useShared){
	/* ha tobb GPU van, akkor az elsot valasztjuk */
	hipSetDevice(0);
	/* kernel parameterek */
	float *device_input, *device_coeffs, *device_output;
	hipMalloc((void**)&device_input,samples*sizeof(float));
	hipMalloc((void**)&device_coeffs,taps*sizeof(float));
	hipMalloc((void**)&device_output,samples*sizeof(float));
	hipMemcpy(device_input,input,samples*sizeof(float),hipMemcpyHostToDevice); 
	hipMemcpy(device_coeffs,coeffs,taps*sizeof(float),hipMemcpyHostToDevice);
	/* ha az L1 cache es a megosztott memoria kozos, akkor a nagyobb megosztott memoriat preferaljuk */
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	/* kernel futtatasa es idomeres */
	dim3 thrBlock(BLOCK_SIZE);
	dim3 thrGrid(samples/BLOCK_SIZE);
	clock_t s0, e0;
	if(useShared){
		s0=clock();
		for(int i=0;i<GPU_RUNS;i++) fir_kernel_sh<<<thrGrid,thrBlock>>>(device_input,device_coeffs,device_output,taps);
		hipDeviceSynchronize();
		e0=clock();
		printf("Execution time with GPU (using shared memory): %.1f us\n",(double)(e0-s0)*1000000.0/(CLOCKS_PER_SEC*GPU_RUNS));
	}
	else{
		s0=clock();
		for(int i=0;i<GPU_RUNS;i++) fir_kernel<<<thrGrid,thrBlock>>>(device_input,device_coeffs,device_output,taps);
		hipDeviceSynchronize();
		e0=clock();
		printf("Execution time with GPU (basic): %.1f us\n",(double)(e0-s0)*1000000.0/(CLOCKS_PER_SEC*GPU_RUNS));
	}
	/* eredmeny masolasa, befejezes */
	hipMemcpy(output,device_output,samples*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(device_input);
	hipFree(device_coeffs); 
	hipFree(device_output);
	hipDeviceReset();
}

int main(){
	/* jel es szuroegyutthatok inicializalasa veletlenszamokkal */
	float *coeffs, *input, *output_cpu, *output_gpu_basic, *output_gpu_shared;
	coeffs=(float*)_aligned_malloc(TAP_NUM*sizeof(float),32);
	input=(float*)_aligned_malloc(SAMPLE_NUM*sizeof(float),32);
	output_cpu=(float*)_aligned_malloc(SAMPLE_NUM*sizeof(float),32);
	output_gpu_basic=(float*)_aligned_malloc(SAMPLE_NUM*sizeof(float),32);
	output_gpu_shared=(float*)_aligned_malloc(SAMPLE_NUM*sizeof(float),32);
	for(int i=0;i<TAP_NUM;i++){
		coeffs[i]=rand()/(float)RAND_MAX;
	}
	for(int i=0;i<SAMPLE_NUM;i++){
		input[i]=rand()/(float)RAND_MAX;
		output_cpu[i]=output_gpu_basic[i]=output_gpu_shared[i]=0.0f;
	}
	/* szures */
	firCPU(input,output_cpu,coeffs,SAMPLE_NUM,TAP_NUM);
	firCUDA(input,output_gpu_basic,coeffs,SAMPLE_NUM,TAP_NUM,false);
	firCUDA(input,output_gpu_shared,coeffs,SAMPLE_NUM,TAP_NUM,true);
	/* ellenorzes */
	double maxerror_basic=0.0, maxerror_shared=0.0;
	for(int i=0;i<SAMPLE_NUM;i++){
		double e=(double)output_cpu[i]-(double)output_gpu_basic[i];
		if(e<0) e=-e;
		if(e>maxerror_basic) maxerror_basic=e;
		e=(double)output_cpu[i]-(double)output_gpu_shared[i];
		if(e<0) e=-e;
		if(e>maxerror_shared) maxerror_shared=e;
	}
	printf("\nMax. error (basic): %.3e\nMax. error (shared memory): %.3e\n",maxerror_basic,maxerror_shared);
	/* befejezes */
	_aligned_free(coeffs);
	_aligned_free(input);
	_aligned_free(output_cpu);
	_aligned_free(output_gpu_basic);
	_aligned_free(output_gpu_shared);
	getchar();
	return 0;
}
