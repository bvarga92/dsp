#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "fir.cuh"

__global__ void fir_kernel(const float* input, const float* coeffs, float* output, int taps){
	int out_addr=blockIdx.x*blockDim.x+threadIdx.x;
	float acc=0.0f;
	#pragma unroll MAC_UNROLL_FACTOR
	for(int tap=taps-1;tap>=0;tap--){
		int in_addr=out_addr-tap;
		if(in_addr>=0) acc=fmaf(coeffs[tap],input[in_addr],acc);
	}
	output[out_addr]=acc;
}

__global__ void fir_kernel_sh(const float* input, const float* coeffs, float* output, int taps){
	int out_addr=blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ float shmem[BLOCK_SIZE];
	shmem[threadIdx.x]=input[out_addr];
	__syncthreads();
	float acc=0.0f;
	#pragma unroll MAC_UNROLL_FACTOR
	for(int tap=taps-1;tap>=0;tap--){
		int in_addr=out_addr-tap;
		if(in_addr>=0){
			if(threadIdx.x>=tap)
				acc=fmaf(coeffs[tap],shmem[threadIdx.x-tap],acc);
			else
				acc=fmaf(coeffs[tap],input[in_addr],acc);
		}
	}
	output[out_addr]=acc;
}
